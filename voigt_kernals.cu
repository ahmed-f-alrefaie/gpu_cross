#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////
/*			VOIGT PROFILE			*/
//////////////////////////////////////////////////////////
__global__ void device_compute_cross_section_voigt_stepone(double* g_energies,const int*  __restrict__ g_gns,const double*  __restrict__ g_nu,const double*  __restrict__ g_aif, const int N_ener){
	//The stored shared data
	
	//Get the global and local thread number
	int g_idx = blockIdx.x * blockDim.x + threadIdx.x;
	double ei,gns,nu_if,aif,abscoef;
	double temp_2 = 1.0;//cross_constants.ln2pi/cross_constants.halfwidth;
	
	//if(g_idx == 0) printf("partition = %12.6f\n",cross_constants.partition);
	if(g_idx < N_ener){
			//Store values in local memory
			ei = g_energies[g_idx];
			gns = g_gns[g_idx];
			nu_if = g_nu[g_idx];
			aif = g_aif[g_idx];
				
			abscoef= cross_constants.cmcoef*temp_2*aif*gns
				*exp(-cross_constants.beta*ei)*(1.0-exp(-cross_constants.beta*nu_if))/
				(nu_if*nu_if*cross_constants.partition);
			if(nu_if==0)abscoef=0.0;
			g_energies[g_idx] = abscoef;
			
	}


}

__global__ void device_compute_cross_section_voigt_stepone(double* g_energies,const int*  g_gns,const double*  g_nu,const double*  g_aif,double*  g_gamma,double*  g_n, const int N_ener){
	//The stored shared data
	
	//Get the global and local thread number
	int g_idx = blockIdx.x * blockDim.x + threadIdx.x;
	double ei,gns,nu_if,aif,abscoef;
	double gammaL;
	
	//cross_constants.ln2pi/cross_constants.halfwidth;
	//if(g_idx == 0) printf("partition = %12.6f\n",cross_constants.partition);
	if(g_idx < N_ener){
			//Store values in local memory
			ei = g_energies[g_idx];
			gns = g_gns[g_idx];
			nu_if = g_nu[g_idx];
			aif = g_aif[g_idx];

			if(nu_if==0) nu_if = 1e-6;
			abscoef= cross_constants.cmcoef*aif*gns
				*exp(-cross_constants.beta*ei)*(1.0-exp(-cross_constants.beta*nu_if))/
				(nu_if*nu_if*cross_constants.partition);
			if(gns==-1) abscoef = aif;
			g_energies[g_idx] = abscoef;

			gammaL = g_gamma[g_idx]*pow(296.0/cross_constants.temperature,g_n[g_idx])*cross_constants.pressure; 
			g_gamma[g_idx] = gammaL;

			//if(threadIdx.x == 0) printf("%14.2E   %14.2E\n",abscoef,gammaL) ;
			
	}


}


__global__ void device_compute_cross_section_voigt_steptwo(const double* g_freq, double* g_cs,const double*  g_nu,const double*  g_abscoef,const int N,const int N_ener,const int start_idx){
	//The stored shared data
	__shared__ double l_nu[VOIGT_SHARED_SIZE];
	__shared__ double l_abscoef[VOIGT_SHARED_SIZE];
	//Get the global and local thread number
	int g_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int l_idx = threadIdx.x;
	int block_dim = VOIGT_SHARED_SIZE;
	double cs_val = 0.0;
	double dfreq_=0.0;
	double freq = 0.0;
	double gammaG=0.05,gammaL=0.05,x,y;
	double dpwcoeff = sqrt(2.0*LN2*BOLTZ*cross_constants.temperature/(cross_constants.mean_mass))/VELLGT;
	//double nu_if;
	//if(g_idx == 0) printf("BLOCK_SIZE = %d\n",blockDim.x);
	if(g_idx < N){
		freq = g_freq[start_idx+g_idx];
		//cs_val = g_cs[start_idx+g_idx];
	}

	//if(g_idx==9999)  printf("%12.6f\n",freq);	

	for(int i = 0; i < N_ener; i+=VOIGT_SHARED_SIZE){
		l_nu[l_idx] = 1.0;
		l_abscoef[l_idx] = 0.0;

		if(i + l_idx < N_ener)
		{	
			l_nu[l_idx] = g_nu[i + l_idx];
			l_abscoef[l_idx] = g_abscoef[i + l_idx];
		}
		
		__syncthreads();
		for(int j = 0; j < VOIGT_SHARED_SIZE; j++){
			dfreq_=l_nu[j]-freq;
			gammaG = l_nu[j]*dpwcoeff;
			x =SQRTLN2*abs(dfreq_)/gammaG;
			y =SQRTLN2*gammaL/gammaG;
			double xxyy = x * x + y * y;

			//Algorithm 916
			if(xxyy < 100.0){
				cs_val+=l_abscoef[j]*SQRTLN2PI/(gammaG)*y*voigt_916(x,y,1.0);					
			}
else{
				//3-point gauss hermite
			cs_val+=l_abscoef[j]*(SQRTLN2PI/gammaG)*voigt_threegausshermite(x,y,xxyy);
			}
			//*__expf(temp_3*dfreq_*dfreq_);

			
		}
		__syncthreads();
		


	}
	

	if(g_idx < N) g_cs[start_idx+g_idx]+=cs_val;


}




__global__ void device_compute_cross_section_voigt_steptwo_block(const double*  g_freq, double* g_cs,const double*   g_nu,const double*  g_abscoef,const int N,const int N_ener,const int start_idx){
	//The stored shared data
	//__shared__ double l_nu[BLOCK_SIZE];
	//__shared__ double l_abscoef[BLOCK_SIZE];
	__shared__ double l_cs_result[VOIGT_BLOCK];
	//Get the global and local thread number
	int b_idx = blockIdx.x;
	int l_idx = threadIdx.x;
	double cs_val = 0.0;
	double dfreq_=0.0;
	double freq = 0.0;
	double nu = 0.0;
	double gammaG=0.05,gammaL=0.05;
	double x,y;

	double dpwcoeff = sqrt(2.0*LN2*BOLTZ*cross_constants.temperature/(cross_constants.mean_mass))/VELLGT;

	//double temp_2=cross_constants.ln2pi/cross_constants.halfwidth;
	//double temp_3 = -cross_constants.ln2*(1.0/(cross_constants.halfwidth*cross_constants.halfwidth));

	freq = g_freq[start_idx + b_idx];
	//cs_val = g_cs[start_idx+g_idx];

	//if(g_idx==9999)  printf("%12.6f\n",freq);	
	l_cs_result[l_idx] = cs_val;
	for(int i = l_idx; i < N_ener; i+=VOIGT_BLOCK){
		nu = 0.0;
		//Read value of nu
		nu = g_nu[i];
		dfreq_ = nu-freq;
		if(dfreq_ < -500.0*gammaL)
			continue;
		if(dfreq_ > 500.0*gammaL)
			break;
		gammaG = nu*dpwcoeff;
		x =SQRTLN2*dfreq_/gammaG;
		y =SQRTLN2*gammaL/gammaG;
		double xxyy = x * x + y * y;

		
		////Algorithm 916
		if(xxyy < 100.0){
			cs_val+=g_abscoef[i]*SQRTLN2PI/(gammaG)*y*voigt_916(x,y,1.0);					
		}else{
			//3-point gauss hermite
			cs_val+=g_abscoef[i]*(SQRTLN2PI/gammaG)*voigt_threegausshermite(x,y,xxyy);
		}
			

	}
	//Store results into shared memory
	l_cs_result[l_idx] = cs_val;
	cs_val = 0;
	//Wait for everone to finish nicely
	__syncthreads();
	if(l_idx == 0){
		for(int i = 0; i < VOIGT_BLOCK; i++)
			cs_val+=l_cs_result[i];
		
		g_cs[start_idx+b_idx]+=cs_val;		
	}

}

__global__ void device_compute_cross_section_voigt_steptwo_block(const double*  g_freq, double* g_cs,const double*   g_nu,const double*  g_abscoef,const double*  g_gamma,const int N,const int N_ener,const int start_idx){
	//The stored shared data
	//__shared__ double l_nu[BLOCK_SIZE];
	//__shared__ double l_abscoef[BLOCK_SIZE];
	__shared__ double l_cs_result[VOIGT_BLOCK];
	//Get the global and local thread number
	int b_idx = blockIdx.x;
	int l_idx = threadIdx.x;
	double cs_val = 0.0;
	double dfreq_=0.0;
	double freq = 0.0;
	double nu = 0.0;
	double gammaG=0.05,gammaL=0.05;
	double x,y;

	double dpwcoeff = sqrt(2.0*BOLTZ*cross_constants.temperature*NA/((cross_constants.mean_mass)))/VELLGT;

	//double temp_2=cross_constants.ln2pi/cross_constants.halfwidth;
	//double temp_3 = -cross_constants.ln2*(1.0/(cross_constants.halfwidth*cross_constants.halfwidth));

	freq = g_freq[start_idx + b_idx];
	//cs_val = g_cs[start_idx+g_idx];

	//if(g_idx==9999)  printf("%12.6f\n",freq);	
	l_cs_result[l_idx] = cs_val;
	for(int i = l_idx; i < N_ener; i+=VOIGT_BLOCK){
		nu = 0.0;
		//Read value of nu
		nu = g_nu[i];
		dfreq_ = nu-freq;
		gammaL = g_gamma[i];
		if(dfreq_ < -500.0*gammaL)
			continue;
		if(dfreq_ > 500.0*gammaL)
			break;
		gammaG = 1.0/(nu*dpwcoeff);
		x =abs(dfreq_)*gammaG;
		y =gammaL*gammaG;
		double xxyy = x * x + y * y;
		double voigt_check;// = voigt_916(x,y,0.9);


		
		////Algorithm 916
		if(xxyy < 100.0){
			voigt_check = voigt_916(x,y,0.9);
			//cs_val+=g_abscoef[i]*voigt_check*gammaG*ISQRTPI;					
		}else if(xxyy < 1.0e6){
			//3-point gauss hermite
			voigt_check = voigt_threegausshermite(x,y,xxyy);
			//cs_val+=g_abscoef[i]*ISQRTPI*gammaG;
		}else{
			voigt_check = y/(PI*xxyy);
			//cs_val+= g_abscoef[i]*ISQRTPI*gammaG;
		}
		cs_val+=g_abscoef[i]*voigt_check*gammaG*ISQRTPI;
		//if((blockIdx.x * blockDim.x + threadIdx.x)==0)  printf("dfreq = %14.4E x=%14.4E y=%14.4E gammaL = %14.4E gammaG = %14.4E abscoef=%14.4E voigt=%14.4E cs_val=%14.4E\n",dfreq_,x,y,gammaL,gammaG,g_abscoef[i],voigt_check,cs_val);			

	}
	//Store results into shared memory
	l_cs_result[l_idx] = cs_val;
	cs_val = 0;
	//Wait for everone to finish nicely
	__syncthreads();
	if(l_idx == 0){
		for(int i = 0; i < VOIGT_BLOCK; i++)
			cs_val+=l_cs_result[i];
		
		g_cs[start_idx+b_idx]+=cs_val;		
	}

}
